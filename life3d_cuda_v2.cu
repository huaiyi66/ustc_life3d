﻿/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号: SA24218133
 * 姓名: 章成胜
 * 邮箱: zhangcs66@mail.ustc.edu.cn
 ------------------------------------------------*/
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <cstdlib> // 提供 atoi

#define AT(x, y, z, N) ((x) * (N) * (N) + (y) * (N) + (z))

using std::cin, std::cout, std::endl;
using std::ifstream, std::ofstream;

// CUDA核函数：模拟一个时间步，利用共享内存优化
__global__ void life3d_kernel(const char *universe, char *next, int N)
{
    // 计算全局坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // 线程边界检查
    if (x >= N || y >= N || z >= N)
        return;

    int idx = AT(x, y, z, N);
    int alive = 0;

    // 计算活邻居数，替换模运算为条件判断
    for (int dx = -1; dx <= 1; dx++)
    {
        int nx = x + dx;
        if (nx < 0)
            nx += N;
        else if (nx >= N)
            nx -= N;
        for (int dy = -1; dy <= 1; dy++)
        {
            int ny = y + dy;
            if (ny < 0)
                ny += N;
            else if (ny >= N)
                ny -= N;
            for (int dz = -1; dz <= 1; dz++)
            {
                int nz = z + dz;
                if (nz < 0)
                    nz += N;
                else if (nz >= N)
                    nz -= N;
                if (dx == 0 && dy == 0 && dz == 0)
                    continue;
                alive += universe[AT(nx, ny, nz, N)];
            }
        }
    }

    // 应用规则
    if (universe[idx] && (alive < 5 || alive > 7))
        next[idx] = 0;
    else if (!universe[idx] && alive == 6)
        next[idx] = 1;
    else
        next[idx] = universe[idx];
}

// 核心模拟函数，将世界向前推进T个时刻（CUDA版本）
void life3d_run_cuda(int N, char *universe, int T)
{
    char *d_universe, *d_next;
    size_t size = N * N * N * sizeof(char);

    // 分配设备内存
    hipMalloc(&d_universe, size);
    hipMalloc(&d_next, size);

    // 复制数据到设备
    hipMemcpy(d_universe, universe, size, hipMemcpyHostToDevice);

    // 定义线程块和网格尺寸
    dim3 blockDim(8, 8, 8); // 每个块512个线程
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y - 1) / blockDim.y,
                 (N + blockDim.z - 1) / blockDim.z);

    for (int t = 0; t < T; t++)
    {
        life3d_kernel<<<gridDim, blockDim>>>(d_universe, d_next, N); // 调用核函数
        hipDeviceSynchronize();                                     // 确保核函数执行完毕

        // 交换指针以避免设备到设备拷贝
        std::swap(d_universe, d_next);
    }

    // 复制结果回主机
    hipMemcpy(universe, d_universe, size, hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_universe);
    hipFree(d_next);
}

void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << endl;
        exit(1);
    }
    file.close();
}

void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

int main(int argc, char **argv)
{
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }

    int N = atoi(argv[1]);
    int T = atoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];

    // 分配主机内存
    char *universe = (char *)malloc(N * N * N);
    if (universe == nullptr)
    {
        cout << "Error: Could not allocate memory for universe." << endl;
        return 1;
    }

    read_file(input_file, universe);

    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();

    life3d_run_cuda(N, universe, T);

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);

    write_file(output_file, universe, N);

    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << (static_cast<double>(T) * N * N * N) / time << endl;

    free(universe);
    return 0;
}
